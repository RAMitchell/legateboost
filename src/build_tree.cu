#include "hip/hip_runtime.h"
/* Copyright 2023 NVIDIA Corporation
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 *
 */
#include "legate_library.h"
#include "legateboost.h"
#include "utils.h"
#include "core/comm/coll.h"
#include "build_tree.h"
#include "cuda_help.h"
#include "kernel_helper.cuh"

#include <thrust/iterator/constant_iterator.h>
#include <thrust/iterator/discard_iterator.h>
#include <thrust/execution_policy.h>

namespace legateboost {

__global__ static void reduce_base_sums(legate::AccessorRO<double, 2> g,
                                        legate::AccessorRO<double, 2> h,
                                        size_t n_local_samples,
                                        int64_t sample_offset,
                                        legate::Buffer<double, 1> base_sums,
                                        size_t n_outputs)
{
  typedef hipcub::BlockReduce<double, THREADS_PER_BLOCK> BlockReduce;
  __shared__ typename BlockReduce::TempStorage temp_storage_g;
  __shared__ typename BlockReduce::TempStorage temp_storage_h;

  int32_t output = blockIdx.y;

  int64_t sample_id = threadIdx.x + blockDim.x * blockIdx.x;

  double G = sample_id < n_local_samples ? g[{sample_id + sample_offset, output}] : 0.0;
  double H = sample_id < n_local_samples ? h[{sample_id + sample_offset, output}] : 0.0;

  double blocksumG = BlockReduce(temp_storage_g).Sum(G);
  double blocksumH = BlockReduce(temp_storage_h).Sum(H);

  if (threadIdx.x == 0) {
    atomicAdd(&base_sums[output], blocksumG);
    atomicAdd(&base_sums[output + n_outputs], blocksumH);
  }
}

template <typename TYPE>
__global__ static void fill_histogram(legate::AccessorRO<TYPE, 2> X,
                                      size_t n_local_samples,
                                      size_t n_features,
                                      int64_t sample_offset,
                                      legate::AccessorRO<double, 2> g,
                                      legate::AccessorRO<double, 2> h,
                                      size_t n_outputs,
                                      legate::AccessorRO<TYPE, 2> split_proposal,
                                      legate::Buffer<int32_t, 1> positions,
                                      legate::Buffer<GPair, 4> histogram,
                                      int64_t depth)
{
  // we assume one block per feature*output selection
  // with each block being 1-dimensional
  int64_t feature = blockIdx.x;
  int64_t output  = blockIdx.y;

  for (int64_t sample_id = threadIdx.x; sample_id < n_local_samples; sample_id += blockDim.x) {
    int32_t sample_pos = positions[sample_id];
    if (sample_pos < 0) continue;
    auto x_value = X[{sample_offset + sample_id, feature}];
    bool left    = x_value <= split_proposal[{depth, feature}];

    int position_in_level = sample_pos - ((1 << depth) - 1);

    // this is probably very slow... we should do this in shared memory per block first maybe
    double* addPosition =
      reinterpret_cast<double*>(&histogram[{position_in_level, feature, output, left}]);
    double tmp = g[{sample_offset + sample_id, output}];
    atomicAdd(addPosition, tmp);
    tmp = h[{sample_offset + sample_id, output}];
    atomicAdd(addPosition + 1, tmp);
  }
}

// Key/value pair to simplify reduction
struct GainFeaturePair {
  double gain;
  int feature;

  __device__ void operator=(const GainFeaturePair& other)
  {
    gain    = other.gain;
    feature = other.feature;
  }

  __device__ bool operator==(const GainFeaturePair& other) const
  {
    return gain == other.gain && feature == other.feature;
  }

  __device__ bool operator>(const GainFeaturePair& other) const { return gain > other.gain; }

  __device__ bool operator<(const GainFeaturePair& other) const { return gain < other.gain; }
};

template <typename TYPE>
__global__ static void perform_best_split(legate::Buffer<GPair, 4> histogram,
                                          size_t n_features,
                                          size_t n_outputs,
                                          legate::AccessorRO<TYPE, 2> split_proposal,
                                          double eps,
                                          double learning_rate,
                                          legate::Buffer<double, 2> tree_leaf_value,
                                          legate::Buffer<double, 2> tree_hessian,
                                          legate::Buffer<int32_t, 1> tree_feature,
                                          legate::Buffer<double, 1> tree_split_value,
                                          legate::Buffer<double, 1> tree_gain,
                                          int64_t depth)
{
  // using one block per (level) node to have blockwise reductions
  int node_id = blockIdx.x;

  typedef hipcub::BlockReduce<GainFeaturePair, THREADS_PER_BLOCK> BlockReduce;
  __shared__ typename BlockReduce::TempStorage temp_storage;

  __shared__ double node_best_gain;
  __shared__ int node_best_feature;

  double thread_best_gain = 0;
  int thread_best_feature = -1;

  for (int feature_id = threadIdx.x; feature_id < n_features; feature_id += blockDim.x) {
    double gain = 0;
    for (int output = 0; output < n_outputs; ++output) {
      auto [G_L, H_L] = histogram[{node_id, feature_id, output, true}];
      auto [G_R, H_R] = histogram[{node_id, feature_id, output, false}];
      auto G          = G_L + G_R;
      auto H          = H_L + H_R;
      if (H_L <= 0.0 || H_R <= 0.0) {
        gain = 0;
        break;
      }
      gain += 0.5 * ((G_L * G_L) / (H_L + eps) + (G_R * G_R) / (H_R + eps) - (G * G) / (H + eps));
    }
    if (gain > thread_best_gain) {
      thread_best_gain    = gain;
      thread_best_feature = feature_id;
    }
  }

  // SYNC BEST GAIN TO FULL BLOCK/NODE
  GainFeaturePair thread_best_pair{thread_best_gain, thread_best_feature};
  GainFeaturePair node_best_pair =
    BlockReduce(temp_storage).Reduce(thread_best_pair, hipcub::Max(), THREADS_PER_BLOCK);
  if (threadIdx.x == 0) {
    node_best_gain    = node_best_pair.gain;
    node_best_feature = node_best_pair.feature;
  }
  __syncthreads();

  // from here on we need the global node id
  if (node_best_gain > eps) {
    int global_node_id = node_id + ((1 << depth) - 1);
    for (int output = threadIdx.x; output < n_outputs; output += blockDim.x) {
      auto [G_L, H_L] = histogram[{node_id, node_best_feature, output, true}];
      auto [G_R, H_R] = histogram[{node_id, node_best_feature, output, false}];

      int left_child                         = global_node_id * 2 + 1;
      int right_child                        = left_child + 1;
      tree_leaf_value[{left_child, output}]  = -(G_L / (H_L + eps)) * learning_rate;
      tree_leaf_value[{right_child, output}] = -(G_R / (H_R + eps)) * learning_rate;
      tree_hessian[{left_child, output}]     = H_L;
      tree_hessian[{right_child, output}]    = H_R;

      if (output == 0) {
        tree_feature[global_node_id]     = node_best_feature;
        tree_split_value[global_node_id] = split_proposal[{depth, node_best_feature}];
        tree_gain[global_node_id]        = node_best_gain;
      }
    }
  }
}

namespace {

void SumAllReduce(legate::TaskContext& context, double* x, int count, hipStream_t stream)
{
  if (context.communicators().size() == 0) return;
  auto& comm            = context.communicators().at(0);
  auto domain           = context.get_launch_domain();
  size_t num_ranks      = domain.get_volume();
  ncclComm_t* nccl_comm = comm.get<ncclComm_t*>();

  if (num_ranks > 1) {
    CHECK_NCCL(ncclAllReduce(x, x, count, ncclDouble, ncclSum, *nccl_comm, stream));
    CHECK_CUDA_STREAM(stream);
  }
}

struct Tree {
  Tree(int max_depth, int num_outputs, hipStream_t stream)
    : num_outputs(num_outputs), max_nodes(1 << (max_depth + 1)), stream(stream)
  {
    leaf_value  = legate::create_buffer<double, 2>({max_nodes, num_outputs});
    feature     = legate::create_buffer<int32_t, 1>({max_nodes});
    split_value = legate::create_buffer<double, 1>({max_nodes});
    gain        = legate::create_buffer<double, 1>({max_nodes});
    hessian     = legate::create_buffer<double, 2>({max_nodes, num_outputs});
  }

  ~Tree()
  {
    leaf_value.destroy();
    feature.destroy();
    split_value.destroy();
    gain.destroy();
    hessian.destroy();
  }

  void InitializeBase(double* base_sums, double learning_rate)
  {
    std::vector<double> base_sums_host(2 * num_outputs);
    CHECK_CUDA(hipMemcpyAsync(base_sums_host.data(),
                               base_sums,
                               sizeof(double) * num_outputs * 2,
                               hipMemcpyDeviceToHost,
                               stream));

    auto exec_policy = thrust::cuda::par.on(stream);
    thrust::fill(
      exec_policy, leaf_value.ptr({0, 0}), leaf_value.ptr({0, 0}) + max_nodes * num_outputs, 0.0);
    thrust::fill(exec_policy, feature.ptr({0}), feature.ptr({0}) + max_nodes, -1);
    thrust::fill(
      exec_policy, hessian.ptr({0, 0}), hessian.ptr({0, 0}) + max_nodes * num_outputs, 0.0);

    CHECK_CUDA(hipStreamSynchronize(stream));

    std::vector<double> leaf_value_init(num_outputs);
    for (auto i = 0; i < num_outputs; ++i) {
      leaf_value_init[i] = (-base_sums_host[i] / base_sums_host[i + num_outputs]) * learning_rate;
    }
    CHECK_CUDA(hipMemcpyAsync(leaf_value.ptr({0, 0}),
                               leaf_value_init.data(),
                               sizeof(double) * num_outputs,
                               hipMemcpyHostToDevice,
                               stream));
    CHECK_CUDA(hipMemcpyAsync(hessian.ptr({0, 0}),
                               base_sums + num_outputs,
                               sizeof(double) * num_outputs,
                               hipMemcpyDeviceToDevice,
                               stream));

    CHECK_CUDA(hipStreamSynchronize(stream));
  }

  template <typename T, int DIM>
  void WriteOutput(legate::Store& out, const legate::Buffer<T, DIM>& x)
  {
    // all outputs are 2D
    // for those where the internal buffer is 1D we expect the 2nd extent to be 1
    const legate::Point<DIM> zero   = legate::Point<DIM>::ZEROES();
    const legate::Point<2> zero2    = legate::Point<2>::ZEROES();
    const legate::Rect<2> out_shape = out.shape<2>();
    auto out_acc                    = out.write_accessor<T, 2>();
    EXPECT(DIM == 2 || out_shape.hi[1] == out_shape.lo[1], "Buffer is 1D but store has 2D.");
    EXPECT(out_shape.lo == zero2, "Output store shape should start at zero.");
    EXPECT(out_acc.accessor.is_dense_row_major(out_shape), "Output store is not dense row major.");
    CHECK_CUDA(hipMemcpyAsync(out_acc.ptr(zero2),
                               x.ptr(zero),
                               out_shape.volume() * sizeof(T),
                               hipMemcpyDeviceToDevice,
                               stream));
  }

  void WriteTreeOutput(legate::TaskContext& context)
  {
    WriteOutput(context.outputs().at(0), leaf_value);
    WriteOutput(context.outputs().at(1), feature);
    WriteOutput(context.outputs().at(2), split_value);
    WriteOutput(context.outputs().at(3), gain);
    WriteOutput(context.outputs().at(4), hessian);
    CHECK_CUDA_STREAM(stream);
  }

  legate::Buffer<double, 2> leaf_value;
  legate::Buffer<int32_t, 1> feature;
  legate::Buffer<double, 1> split_value;
  legate::Buffer<double, 1> gain;
  legate::Buffer<double, 2> hessian;
  const int num_outputs;
  const int max_nodes;
  hipStream_t stream;
};

struct build_tree_fn {
  template <legate::Type::Code CODE>
  void operator()(legate::TaskContext& context)
  {
    using T           = legate::legate_type_of<CODE>;
    const auto& X     = context.inputs().at(0);
    auto X_shape      = X.shape<2>();
    auto X_accessor   = X.read_accessor<T, 2>();
    auto num_features = X_shape.hi[1] - X_shape.lo[1] + 1;
    auto num_rows     = X_shape.hi[0] - X_shape.lo[0] + 1;
    const auto& g     = context.inputs().at(1);
    const auto& h     = context.inputs().at(2);
    EXPECT_AXIS_ALIGNED(0, X.shape<2>(), g.shape<2>());
    EXPECT_AXIS_ALIGNED(0, g.shape<2>(), h.shape<2>());
    EXPECT_AXIS_ALIGNED(1, g.shape<2>(), h.shape<2>());
    auto g_shape                = context.inputs().at(1).shape<2>();
    auto num_outputs            = g.shape<2>().hi[1] - g.shape<2>().lo[1] + 1;
    auto g_accessor             = g.read_accessor<double, 2>();
    auto h_accessor             = h.read_accessor<double, 2>();
    const auto& split_proposals = context.inputs().at(3);
    EXPECT_AXIS_ALIGNED(1, split_proposals.shape<2>(), X.shape<2>());
    auto split_proposal_accessor = split_proposals.read_accessor<T, 2>();

    // Scalars
    auto learning_rate = context.scalars().at(0).value<double>();
    auto max_depth     = context.scalars().at(1).value<int>();
    auto random_seed   = context.scalars().at(2).value<uint64_t>();

    auto stream = legate::cuda::StreamPool::get_stream_pool().get_stream();

    Tree tree(max_depth, num_outputs, stream);

    // Initialize the root node
    {
      // base sums contain g-sums first, h sums second [0,...,num_outputs-1, num_outputs, ...,
      // num_outputs*2 -1]
      auto base_sums = legate::create_buffer<double, 1>(num_outputs * 2);
      CHECK_CUDA(hipMemsetAsync(base_sums.ptr(0), 0, num_outputs * 2 * sizeof(double), stream));

      const size_t blocks = (num_rows + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
      dim3 grid_shape     = dim3(blocks, num_outputs);
      reduce_base_sums<<<grid_shape, THREADS_PER_BLOCK, 0, stream>>>(
        g_accessor, h_accessor, num_rows, X_shape.lo[0], base_sums, num_outputs);
      CHECK_CUDA_STREAM(stream);

      SumAllReduce(context, reinterpret_cast<double*>(base_sums.ptr(0)), num_outputs * 2, stream);

      tree.InitializeBase(base_sums.ptr(0), learning_rate);

      base_sums.destroy();
      CHECK_CUDA_STREAM(stream);
    }

    // Begin building the tree
    auto positions = legate::create_buffer<int32_t, 1>(num_rows);
    CHECK_CUDA(hipMemsetAsync(positions.ptr(0), 0, num_rows * sizeof(int32_t), stream));

    for (int64_t depth = 0; depth < max_depth; ++depth) {
      int max_nodes = 1 << depth;

      // Dimensions[Node, Feature, Output, L/R]
      auto histogram_buffer =
        legate::create_buffer<GPair, 4>({max_nodes, num_features, num_outputs, 2});
      CHECK_CUDA(hipMemsetAsync(histogram_buffer.ptr(legate::Point<4>::ZEROES()),
                                 0,
                                 max_nodes * num_features * num_outputs * 2 * sizeof(GPair),
                                 stream));

      dim3 grid_shape = dim3(num_features, num_outputs);
      fill_histogram<<<grid_shape, THREADS_PER_BLOCK, 0, stream>>>(X_accessor,
                                                                   num_rows,
                                                                   num_features,
                                                                   X_shape.lo[0],
                                                                   g_accessor,
                                                                   h_accessor,
                                                                   num_outputs,
                                                                   split_proposal_accessor,
                                                                   positions,
                                                                   histogram_buffer,
                                                                   depth);
      CHECK_CUDA_STREAM(stream);

      SumAllReduce(context,
                   reinterpret_cast<double*>(histogram_buffer.ptr({0, 0, 0, 0})),
                   max_nodes * num_features * num_outputs * 4,
                   stream);

      // Find the best split
      double eps = 1e-5;
      perform_best_split<<<max_nodes, THREADS_PER_BLOCK, 0, stream>>>(histogram_buffer,
                                                                      num_features,
                                                                      num_outputs,
                                                                      split_proposal_accessor,
                                                                      eps,
                                                                      learning_rate,
                                                                      tree.leaf_value,
                                                                      tree.hessian,
                                                                      tree.feature,
                                                                      tree.split_value,
                                                                      tree.gain,
                                                                      depth);
      CHECK_CUDA_STREAM(stream);

      histogram_buffer.destroy();

      // Update the positions
      auto tree_split_value        = tree.split_value;
      auto tree_feature            = tree.feature;
      auto update_positions_lambda = [=] __device__(size_t idx) {
        int32_t pos = positions[idx];
        if (pos < 0 || tree_feature[pos] == -1) {
          positions[idx] = -1;
          return;
        }
        double x_value = X_accessor[{X_shape.lo[0] + (int64_t)idx, tree_feature[pos]}];
        bool left      = x_value <= tree_split_value[pos];
        positions[idx] = left ? 2 * pos + 1 : 2 * pos + 2;
      };

      LaunchN(num_rows, stream, update_positions_lambda);

      CHECK_CUDA_STREAM(stream);
    }

    if (context.get_task_index()[0] == 0) { tree.WriteTreeOutput(context); }
  }
};

}  // namespace

/*static*/ void BuildTreeTask::gpu_variant(legate::TaskContext& context)
{
  const auto& X = context.inputs().at(0);
  type_dispatch_float(X.code(), build_tree_fn(), context);
}

}  // namespace legateboost
